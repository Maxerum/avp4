#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <Windows.h>
#include <hip/hip_runtime.h> 
#include <intrin.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>

#pragma comment(lib, "cudart") 

//#define SIZE_M 8
//#define SIZE_N 4
//#define COUNT_OF_THREADS 1024
//#define MAX_BLOCKS 200000

#define SIZE_M 64
#define SIZE_N 512
#define GRID_X 16
#define GRID_Y 1
#define BLOCK_X 32
#define BLOCK_Y 32
#define THREAD_ELEMENT_X 1
#define THREAD_ELEMENT_Y 4




using namespace std;

//void cpu_matrixOperation(short*, short*, int, int);
//void cuda_matrixOperation(short*, short*, bool);
//void cuda_checkStatus(hipError_t);
void fillMatrix(int*, int, int);
bool checkEquality(int*, int*, int, int);

void showMatrix(int* mat, int sizeOfM, int sizeOfN) {
	for (int i = 0; i < sizeOfN / 4; i++) {
		for (int j = 0; j < sizeOfM /8; j++) {
			cout << mat[(sizeOfM / 2) * i + j] << " ";
		}
		cout << endl;
	}
	/*for (int i = 0; i < size; i++) {

		cout << mat[i] << endl;
	}*/
}

void cpuMatrix(int* inMatrix, int* outMatrix, int sizeOfM, int sizeOfN) {
	/*clock_t startTime, endTime;
	startTime = clock();*/
	int orfer[] = { 1, 2 , 0 ,3 };
	//sizeOfM >> sizeOfN
	LARGE_INTEGER frequency, start, finish;
	float delay;
	QueryPerformanceFrequency(&frequency);

	QueryPerformanceCounter(&start);

	for (int h = 0; h < sizeOfM; h += 4) {
		for (auto i = 0; i < sizeOfN; i++) {

			for (auto j = h, counter = 0; counter < 4; j++, counter++) {
				int tmp = orfer[counter];
				int a = (j + 1) % 2 == 0 ? 1 : 0;
				if (h > 0) {

					a += (h/4) * 2;
					tmp = orfer[counter] + h;
				}
				//cout << "INndexOUT " << (counter / 2) * sizeOfM / 2 + a + i * 2 * sizeOfM / 2 << " INDEXIN " << i * sizeOfM + tmp << endl;
				outMatrix[(counter / 2) * sizeOfM / 2 + a + i * 2 * sizeOfM / 2] = inMatrix[i * sizeOfM + tmp];
				//cout << outMatrix[(counter / 2) * sizeOfM / 2 + a + i * 2 * sizeOfM / 2] << " " << inMatrix[i * sizeOfM + tmp] << endl;
			}
		}
	}
	QueryPerformanceCounter(&finish);
	delay = (finish.QuadPart - start.QuadPart) * 1000.0f / frequency.QuadPart;
	printf("The time for cpu spend: %.3f ms\n", delay);
}


__global__ void cudaKernel(int *init, int* dest) {
	//const int offsetX = BLOCK_X * blockIdx.x  + threadIdx.x;
	const int offsetX = 4 * blockIdx.x + threadIdx.x;
	const int offsetY = threadIdx.y;
	//const int offsetY = BLOCK_Y * blockIdx.y * THREAD_ELEMENT_Y + threadIdx.y;

	int a = init[offsetY * SIZE_M + offsetX + 0];
	int b = init[offsetY * SIZE_M + offsetX + 1];
	int c = init[offsetY * SIZE_M + offsetX + 2];
	int d = init[offsetY * SIZE_M + offsetX + 3];

	
	const int offsetOutX = 2 * blockIdx.x + threadIdx.x;
	const int offsetOutY = 2 * threadIdx.y;

	dest[offsetOutY * SIZE_M/2 + offsetOutX + 0] = b;
	dest[offsetOutY * SIZE_M/2  + offsetOutX + 1] = c;
	dest[offsetOutY * SIZE_M/2  + SIZE_M /2 + offsetOutX + 1] = d;
	dest[offsetOutY * SIZE_M/2  + SIZE_M /2 + offsetOutX + 0] = a;
	
}

void showPartOfMatrix(int *matrix) {
	for (int i = 0; i < SIZE_N / 4; i++) {
		for (int j = 0; j < SIZE_M / 4; j++) {
			cout << matrix[i * SIZE_M + j] << " ";
		}
		cout << endl;
	}
}

void cudaMatrix(int *init, int *dest) {
	float resultTime;

	int* deviceInMatrix;
	int* deviceOutMatrix;
	//события для замера времени в CUDA

	hipEvent_t cuda_startTime;
	hipEvent_t cuda_endTime;
	//Создание событий
	hipEventCreate(&cuda_startTime);
	hipEventCreate(&cuda_endTime);

	hipMalloc(&deviceInMatrix, (SIZE_M*SIZE_N * sizeof(int)));
	hipMalloc(&deviceOutMatrix, (SIZE_N * SIZE_M * sizeof(int)));
	hipMemcpy(deviceInMatrix, init, SIZE_M * SIZE_N * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(16, 1);
	dim3 dimBlock(1, 512);

	//? вопросов много
	/*int row_len = GRID_X * BLOCK_X * THREAD_ELEMENT_X;*/
	hipEventRecord(cuda_startTime, 0);


	cudaKernel << <dimGrid, dimBlock >> > (deviceInMatrix, deviceOutMatrix);



	hipPeekAtLastError();
	hipDeviceSynchronize();
	hipEventRecord(cuda_endTime, 0);
	hipEventSynchronize(cuda_endTime);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, cuda_startTime, cuda_endTime);

	hipEventDestroy(cuda_startTime);
	hipEventDestroy(cuda_endTime);

	printf("The time for cuda with global memory spend: %.3f ms\n", elapsedTime);

	hipMemcpy(init, deviceInMatrix, SIZE_N*SIZE_M * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dest, deviceOutMatrix, SIZE_N*SIZE_M * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(deviceInMatrix);
	hipFree(deviceOutMatrix);
}

int main() {
	//выделение памяти под матрицы в cpu
	int* initMatrix = (int*)malloc(SIZE_M * SIZE_N * sizeof(int));
	int* cpu_outMatrix = (int*)malloc(SIZE_M * SIZE_N * sizeof(int));
	int* cuda_outMatrix = (int*)malloc(SIZE_M * SIZE_N * sizeof(int));
	int* cuda_outMatrixSharedMemory = (int*)malloc(SIZE_M * SIZE_N * sizeof(int));

	fillMatrix(initMatrix, SIZE_M, SIZE_N);
	//showMatrix(initMatrix, SIZE_M , SIZE_N);
	/*cuda_matrixOperation(initMatrix, cuda_outMatrix, false);
	cuda_matrixOperation(initMatrix, cuda_outMatrixSharedMemory, true);*/

	/*cpu_matrixOperation(initMatrix, cpu_outMatrix, SIZE_M, SIZE_N);*/



	cpuMatrix(initMatrix, cpu_outMatrix, SIZE_M, SIZE_N);
	//showPartOfMatrix(cpu_outMatrix);
	showMatrix(cpu_outMatrix, SIZE_M, SIZE_N);

	cudaMatrix(initMatrix, cuda_outMatrix);
	//showPartOfMatrix(cuda_outMatrix);

	showMatrix(cuda_outMatrix, SIZE_M, SIZE_N);

	//showMatrix(cpu_outMatrix, SIZE_M/2 , SIZE_N * 2);
	if (checkEquality(cuda_outMatrix, cpu_outMatrix, SIZE_M, SIZE_N)) {
		cout << "Results are equals!" << endl;
	}
	else {
		cout << "Results are NOT equals!" << endl;
	}


	free(initMatrix);
	free(cpu_outMatrix);
	free(cuda_outMatrix);
	free(cuda_outMatrixSharedMemory);
}


//global - ключевое слово, предназначено для указания како
//ядро - функция, которая описывает последовательность операций , которые выполнятся на каждой нити параллельно
//__global__ void cuda_matrixSharedMemoryOperationKernel(int* inMatrix, int* outMatrix, int numOfBlocksInRow) {
//	int remainderElements = SIZE_M % COUNT_OF_THREADS;
//
//	__shared__ int sharedMemory[COUNT_OF_THREADS];
//	__shared__ short sharedMemoryOut[COUNT_OF_THREADS * 2];
//
//	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0 && threadIdx.x >= remainderElements) {
//		return;
//	}
//
//	int *startOfResultRow = &inMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow)];
//	outMatrix = &outMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow)];
//
//	int countOfThreads = 0;
//
//	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0) {
//		countOfThreads = remainderElements;
//	}
//	else {
//		countOfThreads = COUNT_OF_THREADS;
//	}
//
//	if (threadIdx.x < (countOfThreads / 2)) {
//		sharedMemory[threadIdx.x] = startOfResultRow[(blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2 + threadIdx.x];
//	}
//	else {
//		sharedMemory[threadIdx.x] = startOfResultRow[threadIdx.x % (countOfThreads / 2) + SIZE_M / 2 + (blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2];
//	}
//
//	int elements = sharedMemory[threadIdx.x];
//	short firstElement = (short)elements;
//	short secondElement = (short)(elements >> 16);
//
//	int offset = COUNT_OF_THREADS * 2 * (blockIdx.x % numOfBlocksInRow);
//
//	if (threadIdx.x < (countOfThreads / 2)) {
//		sharedMemoryOut[threadIdx.x * 2 * 2] = firstElement;
//		sharedMemoryOut[(threadIdx.x * 2 + 1) * 2] = secondElement;
//	}
//	else {
//		sharedMemoryOut[(threadIdx.x - countOfThreads / 2) * 2 * 2 + 1] = firstElement;
//		sharedMemoryOut[((threadIdx.x - countOfThreads / 2) * 2 + 1) * 2 + 1] = secondElement;
//	}
//
//	__syncthreads();
//
//	outMatrix[offset / 2 + threadIdx.x] = ((int*)sharedMemoryOut)[threadIdx.x];
//}



void cpu_matrixOperation(short* inMatrix, short* outMatrix, int sizeOfM, int sizeOfN) {
	clock_t startTime, endTime;
	startTime = clock();
	//sizeOfM >> sizeOfN
	for (auto i = 0; i < sizeOfM; i++) {
		for (auto j = 0; j < sizeOfN; j++) {
			int a = (j + 1) % 2 == 0 ? 1 : 0;
			outMatrix[(j / 2) * sizeOfM * 2 + a + i * 2] = inMatrix[i + sizeOfM * j];
		}
	}
	endTime = clock();
	printf("CPU time: %lf seconds\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);
}



void fillMatrix(int* matrix, int sizeM, int sizeN)
{
	int counter = 0;
	for (int i = 0; i < sizeN; ++i)
	{
		for (int j = 0; j < sizeM; ++j)
		{
			matrix[sizeM * i + j] = counter++;
		}
	}
}

//void cuda_checkStatus(hipError_t cudaStatus) {
//	if (cudaStatus != hipSuccess) {
//		cout << "CUDA return error code: " << cudaStatus;
//		cout << " " << hipGetErrorString(cudaStatus) << endl;
//		exit(-1);
//	}
//}

bool checkEquality(int* inMatrix, int* outMatrix, int sizeOfM, int sizeOfN) {
	for (int i = 0; i < sizeOfN * sizeOfM; i++) {
		if (inMatrix[i] != outMatrix[i]) {
			return false;
		}
	}
	return true;
}
